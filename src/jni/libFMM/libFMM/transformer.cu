#include "hip/hip_runtime.h"

#include "transformer.h"

#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <>

__global__ void multiply_matrix(const float m1[4][4], const float m2[4][4], float m[4][4])
{
	__shared__ float assist[4][4][4];
	assist[threadIdx.x][threadIdx.y][threadIdx.z] = m1[threadIdx.z][threadIdx.y] * m2[threadIdx.x][threadIdx.z];

	__syncthreads();
	if (threadIdx.z < 2) assist[threadIdx.x][threadIdx.y][threadIdx.z] += assist[threadIdx.x][threadIdx.y][threadIdx.z + 2];
	__syncthreads();
	if (threadIdx.z < 1) m[threadIdx.x][threadIdx.y] = assist[threadIdx.x][threadIdx.y][0] + assist[threadIdx.x][threadIdx.y][1];
}

void MultiplyMatrix(const float m1[4][4], const float m2[4][4], float m[4][4])
{
	const static dim3 blockSize(4, 4, 4);
	multiply_matrix<<<1, blockSize>>>(m1, m2, m);
}

__global__ void calculate_transform_matrix(float offx, float offy, float offz, float rotx, float roty, float rotz, float matrix[4][4])
{
	
}

__device__ void transform_coordinate(float *dstcrd, const float *srccrd, float matrix[4][4])
{
	__shared__ float assist_matrix[4][4];
	assist_matrix[threadIdx.y][threadIdx.x] = threadIdx.y != 3 ? matrix[threadIdx.y][threadIdx.x] * srccrd[threadIdx.y] : matrix[threadIdx.y][threadIdx.x];
	__syncthreads();
	if (threadIdx.y < 2) assist_matrix[threadIdx.y][threadIdx.x] += assist_matrix[threadIdx.y + 2][threadIdx.x];
	__syncthreads();
	if (threadIdx.y < 1) dstcrd[threadIdx.x] = assist_matrix[0][threadIdx.x] + assist_matrix[1][threadIdx.x];
}

__global__  void transform_coordinate(float (*dstcrd)[3], const float (*srccrd)[3], float matrix[4][4], unsigned int crdnum)
{
	unsigned int crdidx = blockIdx.x;
	while (crdidx < crdnum)
	{
		transform_coordinate(dstcrd[crdidx], srccrd[crdidx], matrix);
		crdidx += gridDim.x;
	}
}

void TransformCoordinate(float (*dstCrd)[3], const float (*srcCrd)[3], float matrix[4][4], unsigned int crdNum)
{
	const static dim3 blockSize(3, 4);
	transform_coordinate << <1, blockSize >> > (dstCrd, srcCrd, matrix, crdNum);
}
